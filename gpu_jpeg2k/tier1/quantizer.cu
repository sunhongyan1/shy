#include "hip/hip_runtime.h"
/* 
Copyright 2009-2013 Poznan Supercomputing and Networking Center

Authors:
Milosz Ciznicki miloszc@man.poznan.pl

GPU JPEG2K is free software: you can redistribute it and/or modify
it under the terms of the GNU Affero General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.

GPU JPEG2K is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE. See the
GNU Affero General Public License for more details.

You should have received a copy of the GNU Affero General Public License
along with GPU JPEG2K. If not, see <http://www.gnu.org/licenses/>.
*/
/**
 * @file quantization.cu
 *
 * @author Milosz Ciznicki
 * @date 06-09-2010
 */
#include<stdio.h>
#include "../types/image_types.h"
#include "../misc/cuda_errors.h"
extern "C" {
	#include "quantizer.h"
	#include "quantization.h"
	#include "../misc/memory_management.cuh"
	#include "../print_info/print_info.h"
}

/**
 * @brief Subband quantization.
 *
 * @param idata Input tile_comp_data.
 * @param size Width and height of subbnad.
 * @param step_size Step size(deltab).
 */
__global__
void subband_quantization_lossy(type_data *idata, int2 isize, int *odata, int2 osize, int2 cblk_size, const float convert_factor)
{
	int cblk_sum_size = cblk_size.x * cblk_size.y;
	int block_id = blockIdx.x + blockIdx.y * gridDim.x;
	int i = threadIdx.x;
	int j = threadIdx.y;
	int n = i + blockIdx.x * cblk_size.x;
	int m = j + blockIdx.y * cblk_size.y;
	int in = n + m * isize.x;
	int out = i + j * cblk_size.x + (block_id) * cblk_sum_size;
	int cache_value;

	while(j < cblk_size.y && m < osize.y)
	{
		while(i < cblk_size.x &&  n < osize.x)
		{
			cache_value = idata[in] * convert_factor;
			odata[out] = cache_value < 0 ? (1 << 31) | ((int)(-cache_value)) : cache_value;
//			odata[out] = (idata[in] * convert_factor) < 0 ? (1 << 31) | ((int)(-(idata[in] * convert_factor))) : (idata[in] * convert_factor);
			i += BLOCKSIZEX;
			n = i + blockIdx.x * cblk_size.x;
			in = n + m * isize.x;
			out = i + j * cblk_size.x + (block_id) * cblk_sum_size;
		}
		i = threadIdx.x;
		j += BLOCKSIZEY;
		n = i + blockIdx.x * cblk_size.x;
		m = j + blockIdx.y * cblk_size.y;
		in = n + m * isize.x;
		out = i + j * cblk_size.x + (block_id) * cblk_sum_size;
	}
}

/**
 * @brief Subband quantization.
 *
 * @param idata Input tile_comp_data.
 * @param size Width and height of subbnad.
 * @param step_size Step size(deltab).
 */
__global__
void subband_quantization_lossless(type_data *idata, int2 isize, int *odata, int2 osize, int2 cblk_size, const int shift_bits)
{
	int cblk_sum_size = cblk_size.x * cblk_size.y;
	int block_id = blockIdx.x + blockIdx.y * gridDim.x;
	int i = threadIdx.x;
	int j = threadIdx.y;
	int n = i + blockIdx.x * cblk_size.x;
	int m = j + blockIdx.y * cblk_size.y;
	int in = n + m * isize.x;
	int out = i + j * cblk_size.x + (block_id) * cblk_sum_size;
	int cache_value;

	while(j < cblk_size.y && m < osize.y)
	{
		while(i < cblk_size.x &&  n < osize.x)
		{
			cache_value = ((int)idata[in]) << shift_bits;
			odata[out] = cache_value < 0 ? (1 << 31) | (-cache_value) : cache_value;
//			odata[out] = (idata[in] * convert_factor) < 0 ? (1 << 31) | ((int)(-(idata[in] * convert_factor))) : (idata[in] * convert_factor);
			i += BLOCKSIZEX;
			n = i + blockIdx.x * cblk_size.x;
			in = n + m * isize.x;
			out = i + j * cblk_size.x + (block_id) * cblk_sum_size;
		}
		i = threadIdx.x;
		j += BLOCKSIZEY;
		n = i + blockIdx.x * cblk_size.x;
		m = j + blockIdx.y * cblk_size.y;
		in = n + m * isize.x;
		out = i + j * cblk_size.x + (block_id) * cblk_sum_size;
	}
}

/**
 * @brief Subband quantization.
 *
 * @param subband Input subband.
 * @param size Width and height of subbnad.
 */
type_subband *quantization(type_subband *sb)
{
//	println_start(INFO);

	dim3 blocks;
	dim3 threads;
	int i;
	type_codeblock *cblk;

	type_res_lvl *res_lvl = sb->parent_res_lvl;
	type_tile_comp *tile_comp = res_lvl->parent_tile_comp;
	type_image *img = tile_comp->parent_tile->parent_img;

	/* Base step size */
	float base_step = /*BASE_STEP*/(float)1 / (float)(1 << (img->num_range_bits - 1));
	int guard_bits = GUARD_BITS;
	/* Relative quantization step size. Step size is signaled relative to the wavelet coefficient bit depth. */
	float relative_step_size;
	int shift_bits;
       // FILE *fp=fopen("number5","a+");
	/* Lossy */
	if(img->wavelet_type)
	{
		/* The number of magnitude bits in the integer representation of the quantized data */
		sb->mag_bits = guard_bits - 1 + res_lvl->dec_lvl_no - (int)(log(base_step) / log(2));
		/* Relative quantization step size. Step size is signaled relative to the wavelet coefficient bit depth. */
		relative_step_size = base_step / (1 << res_lvl->dec_lvl_no);
		shift_bits = 31 - sb->mag_bits;

		/* Calculate step size and convert factor for quantization */
		relative_step_size = convert_from_exp_mantissa(convert_to_exp_mantissa(relative_step_size));
		sb->step_size = ((1L << (img->num_range_bits + get_exp_subband_gain(sb->orient))) * relative_step_size);
		sb->convert_factor = (1.0f / sb->step_size) * (1 << (shift_bits));

		//println_var(INFO, "Lossy mag_bits:%d convert_factor:%f shift_bits:%d relative_step_size:%f step_size:%f subband_gain:%d", sb->mag_bits, sb->convert_factor, shift_bits, relative_step_size, sb->step_size, get_exp_subband_gain(sb->orient));

	} else /* Lossless */
	{
		/* K_b_max = B - 1 + X_b + G */
		sb->mag_bits = img->num_range_bits - 1 + get_exp_subband_gain(sb->orient) + guard_bits;
		shift_bits = 31 - sb->mag_bits;
//fprintf(fp,"%d ",shift_bits);
		//printf("magbits %d\n", sb->mag_bits);
		sb->convert_factor = 1 << shift_bits;

		//println_var(INFO, "Lossless mag_bits:%d convert_factor:%f shift_bits:%d subband_gain:%d", sb->mag_bits, sb->convert_factor, shift_bits, get_exp_subband_gain(sb->orient));
	}
//fclose(fp);
	//println_var(INFO, "%d %d %d %d %d %d %d", sb->width,sb->height, sb->num_cblks, sb->num_xcblks, sb->num_ycblks, tile_comp->cblk_w, tile_comp->cblk_h);
	/* Allocate aligned memory for linear codeblocks data */
	cuda_d_allocate_mem((void **) &(sb->cblks_data_d), sb->num_cblks * tile_comp->cblk_w * tile_comp->cblk_h * sizeof(int));

	/* Input and output data */
	type_data *idata = tile_comp->img_data_d + sb->tlx + sb->tly * tile_comp->width;
	int *odata = sb->cblks_data_d;
	int2 isize = make_int2(tile_comp->width, tile_comp->height);
	int2 osize = make_int2(sb->width, sb->height);
	int2 cblk_size = make_int2(tile_comp->cblk_w, tile_comp->cblk_h);

	/* Number of blocks for parallel reduction */
	blocks = dim3(sb->num_xcblks , sb->num_ycblks);
	/* Number of threads for parallel reduction */
	threads = dim3(BLOCKSIZEX, BLOCKSIZEY);

	//println_var(INFO, "blocks.x:%d blocks.y:%d threads.x:%d threads.y:%d", blocks.x, blocks.y, threads.x, threads.y);
	//println_var(INFO, "isize.x:%d isize.y:%d osize.x:%d osize.y:%d cblk_size.x:%d cblk_size.y:%d", isize.x, isize.y, osize.x, osize.y, cblk_size.x, cblk_size.y);
	if(img->wavelet_type)
	{
		subband_quantization_lossy<<<blocks, threads>>>(idata, isize, odata, osize, cblk_size, sb->convert_factor);
	} else
	{
		subband_quantization_lossless<<<blocks, threads>>>(idata, isize, odata, osize, cblk_size, shift_bits);
	}

	hipDeviceSynchronize();


	checkCUDAError("quantization");
//printf("the quantier is begin");
	/* Update code blocks pointer to data */
	for(i = 0; i < sb->num_cblks; i++)
	{
		cblk = &(sb->cblks[i]);
		cblk->data_d = sb->cblks_data_d + cblk->cblk_no * tile_comp->cblk_w * tile_comp->cblk_h;
#if 0
		{
				int xxx = 0;
				int32_t *pHost;
				int cblk_size = tile_comp->cblk_w * tile_comp->cblk_h;
				printf("tile_comp->cblk_w=%d , tile_comp->cblk_h=%d , cblk_size=%d\n",tile_comp->cblk_w,tile_comp->cblk_h,cblk_size);
				(pHost) = (int32_t *)malloc(cblk_size*sizeof(int32_t));
				hipMemcpy(pHost,cblk->data_d,cblk_size*sizeof(int32_t),hipMemcpyDeviceToHost);
				hipDeviceSynchronize();
				for(xxx=0;xxx<128;xxx++)
				{
					printf("%012d,", pHost[xxx]);
					if(xxx%16 == 15)
					{
						printf("\n");
					}
				}
				int iend = tile_comp->cblk_w * tile_comp->cblk_h-128;
				printf("iend = %d\n",iend);
				for(xxx=0;xxx<128;xxx++)
				{
					printf("%012d,", pHost[iend+xxx]);
					if(xxx%16 == 15)
					{
						printf("\n");
					}
				}
				printf("\n");
				free(pHost);
		}
#endif
	}
//printf("the quantier is end");
//	println_end(INFO);

	return sb;
}

/**
 * @brief Do quantization for every subbands from tile.
 * @param tile
 */
void quantize_tile(type_tile *tile)
{
//	println_start(INFO);

//	start_measure();
FILE *fp1=fopen("./the_analysic_data/the_quantized_data","w+");
	type_image *img = tile->parent_img;
	type_tile_comp *tile_comp;
	type_res_lvl *res_lvl;
	type_subband *sb;
type_codeblock *cblk;
	int i, j, k;
//fprintf(fp1,"img->num_components=%d \n",img->num_components);
	for(i = 0; i < img->num_components; i++)
	{
   
		tile_comp = &(tile->tile_comp[i]);
   // fprintf(fp1,"the num_rlvls of  tile_comp[%d] is %d \n",i,tile_comp->num_rlvls);
		for(j = 0; j < tile_comp->num_rlvls; j++)
		{
			res_lvl = &(tile_comp->res_lvls[j]);
			//printf("tile_comp->num_rlvls:%d,res_lvl->num_subbands:%d\n",tile_comp->num_rlvls,res_lvl->num_subbands);
                           //  fprintf(fp1,"the num_subbands of  tile_comp[%d]'s num_rlvls[%d] is %d \n",i,j,res_lvl->num_subbands);
			for(k = 0; k < res_lvl->num_subbands; k++)
			{
                                //printf("the max size is %d\n",img->num_components*tile_comp->num_rlvls*res_lvl->num_subbands);
				sb = &(res_lvl->subbands[k]);
 // fprintf(fp1,"the num_subbands of  tile_comp[%d]'s num_rlvls[%d] res_lvl->subbands[%d] is %d \n",i,j,k,sb->num_cblks);
	/* Update code blocks pointer to data */

				quantization(sb);
//cuda_d_allocate_mem((void **) &(sb->cblks_data_d), sb->num_cblks * tile_comp->cblk_w * tile_comp->cblk_h * sizeof(int));
	/*for(int m = 0; m < sb->num_cblks; m++)
	{
		cblk = &(sb->cblks[m]);
fprintf(fp1,"the cblks index is %d\n",m);
		//cblk->data_d = sb->cblks_data_d + cblk->cblk_no * tile_comp->cblk_w * tile_comp->cblk_h;
#if 1
              //   printf("sdadsasdsad\n");
		{
				int xxx = 0;
				int32_t *pHost;
				int cblk_size = tile_comp->cblk_w * tile_comp->cblk_h;
				//printf("tile_comp->cblk_w=%d , tile_comp->cblk_h=%d , cblk_size=%d\n",tile_comp->cblk_w,tile_comp->cblk_h,cblk_size);
                            	(pHost) = (int32_t *)malloc(cblk_size*sizeof(int32_t));
				hipMemcpy(pHost,cblk->data_d,cblk_size*sizeof(int32_t),hipMemcpyDeviceToHost);
				hipDeviceSynchronize();
                                
				for(xxx=0;xxx<16;xxx++)
				{
					//printf("%006d ", pHost[xxx]);
                                        fprintf(fp1,"%006d ",pHost[xxx]);
                                        fprintf(fp1,"%006x ",pHost[xxx]);
					if(xxx%16 == 15)
					{
						//printf("\n");
                                               fprintf(fp1,"\n");
					}
				}
				free(pHost);
		}
#endif
	}*/


			}
		}
		cuda_d_free(tile_comp->img_data_d);
	}
//fclose(fp1);
//fp1=NULL;
//	stop_measure(INFO);

//	println_end(INFO);
}

/**
 * @brief Do quantization for every subbands from tile.
 * @param tile
 */
/*void quantize_tile_dbg(type_tile *tile)
{
	println_start(INFO);

	start_measure();

	type_image *img = tile->parent_img;
	type_tile_comp *tile_comp;
	type_res_lvl *res_lvl;
	type_subband *sb;
	int i, j, k;

	int x = 0, y = 0;
	int size = tile->width * tile->height * sizeof(type_data);
	type_data *buff = (type_data*)malloc(size);

	for(i = 0; i < img->num_components; i++)
	{
		tile_comp = &(tile->tile_comp[i]);
		cuda_memcpy_dth(tile_comp->img_data_d, buff, size);

		for(y = 0; y < tile_comp->height; y++)
		{
			for(x = 0; x < tile_comp->width; x++)
			{
				printf("%f, ", buff[x + y * tile_comp->width]);
			}
		}
		printf("\n");

		for(j = 0; j < tile_comp->num_rlvls; j++)
		{
			res_lvl = &(tile_comp->res_lvls[j]);
			for(k = 0; k < res_lvl->num_subbands; k++)
			{
				sb = &(res_lvl->subbands[k]);
				quantization(sb);
			}
		}
		cuda_d_free(tile_comp->img_data_d);
	}

	stop_measure(INFO);

	println_end(INFO);
}*/

/**
 * @brief Subband quantization.
 *
 * @param subband Input subband.
 * @param size Width and height of subbnad.
 */
//type_subband *quantization_dbg(type_subband *sb)
//{
////	println_start(INFO);
//
//	dim3 blocks;
//	dim3 threads;
//	int i;
//	type_codeblock *cblk;
//
//	type_res_lvl *res_lvl = sb->parent_res_lvl;
//	type_tile_comp *tile_comp = res_lvl->parent_tile_comp;
//	type_image *img = tile_comp->parent_tile->parent_img;
//
//	/* Base step size */
//	float base_step = BASE_STEP;
//	int guard_bits = GUARD_BITS;
//	/* Relative quantization step size. Step size is signaled relative to the wavelet coefficient bit depth. */
//	float relative_step_size;
//	int shift_bits;
//
//	/* Lossy */
//	if(img->wavelet_type)
//	{
//		/* The number of magnitude bits in the integer representation of the quantized data */
//		sb->mag_bits = guard_bits - 1 + res_lvl->dec_lvl_no - (int)(log(base_step) / log(2));
//		/* Relative quantization step size. Step size is signaled relative to the wavelet coefficient bit depth. */
//		relative_step_size = base_step / (1 << res_lvl->dec_lvl_no);
//		shift_bits = 31 - sb->mag_bits;
//
//		/* Calculate step size and convert factor for quantization */
//		relative_step_size = convert_from_exp_mantissa(convert_to_exp_mantissa(relative_step_size));
//		sb->step_size = ((1L << (img->nom_range_bits + get_exp_subband_gain(sb->orient))) * relative_step_size);
//		sb->convert_factor = (1.0f / sb->step_size) * (1 << (shift_bits));
//
////		println_var(INFO, "Lossy mag_bits:%d convert_factor:%f shift_bits:%d relative_step_size:%f step_size:%f subband_gain:%d", sb->mag_bits, sb->convert_factor, shift_bits, relative_step_size, sb->step_size, get_exp_subband_gain(sb->orient));
//
//	} else /* Lossless */
//	{
//		/* K_b_max = B - 1 + X_b + G */
//		sb->mag_bits = img->nom_range_bits - 1 + get_exp_subband_gain(sb->orient) + guard_bits;
//		shift_bits = 31 - sb->mag_bits;
//		sb->convert_factor = 1 << shift_bits;
//
////		println_var(INFO, "Lossless mag_bits:%d convert_factor:%f shift_bits:%d subband_gain:%d", sb->mag_bits, sb->convert_factor, shift_bits, get_exp_subband_gain(sb->orient));
//	}
////	println_var(INFO, "%d %d %d %d %d %d", sb->width, sb->num_cblks, tile_comp->cblk_w, tile_comp->cblk_h, sizeof(long int), sizeof(size_t));
//	/* Allocate aligned memory for linear codeblocks data */
//	cuda_d_allocate_mem((void **) &(sb->cblks_data_d), sb->num_cblks * tile_comp->cblk_w * tile_comp->cblk_h * sizeof(int));
//
//	/* Input and output data */
//	type_data *idata = tile_comp->img_data_d + sb->tlx + sb->tly * tile_comp->height;
//	int *odata = sb->cblks_data_d;
//	int2 isize = make_int2(tile_comp->width, tile_comp->height);
//	int2 osize = make_int2(sb->width, sb->height);
//	int2 cblk_size = make_int2(tile_comp->cblk_w, tile_comp->cblk_h);
//
//	/* Number of blocks for parallel reduction */
//	blocks = dim3(sb->num_xcblks , sb->num_ycblks);
//	/* Number of threads for parallel reduction */
//	threads = dim3(BLOCKSIZEX, BLOCKSIZEY);
//
////	quantization_dbg(idata, osize.x * osize.y * sizeof(type_data), osize.x, osize.y);
//
////	println_var(INFO, "blocks.x:%d blocks.y:%d threads.x:%d threads.y:%d", blocks.x, blocks.y, threads.x, threads.y);
////	println_var(INFO, "isize.x:%d isize.y:%d osize.x:%d osize.y:%d cblk_size.x:%d cblk_size.y:%d", isize.x, isize.y, osize.x, osize.y, cblk_size.x, cblk_size.y);
//	if(img->wavelet_type)
//	{
//		subband_quantization_lossy<<<blocks, threads>>>(idata, isize, odata, osize, cblk_size, sb->convert_factor);
//	} else
//	{
//		subband_quantization_lossless<<<blocks, threads>>>(idata, isize, odata, osize, cblk_size, shift_bits);
//	}
//
//	hipDeviceSynchronize();
//
//	checkCUDAError("quantization");
//
//	int size = tile_comp->cblk_w * tile_comp->cblk_h * sizeof(int);
//	int *buff = (int*)malloc(size);
//	int x = 0, y = 0;
//
//	/* Update code blocks pointer to data */
//	for(i = 0; i < sb->num_cblks; i++)
//	{
//		cblk = &(sb->cblks[i]);
//		cblk->data_d = sb->cblks_data_d + cblk->cblk_no * tile_comp->cblk_w * tile_comp->cblk_h;
//
//		cuda_memcpy_dth(cblk->data_d, buff, size);
//
//		printf("ulx:%d uly:%d magbits:%d orient:%d con:%f\n", sb->tlx, sb->tly, sb->mag_bits, sb->orient, sb->convert_factor);
//
//		for(y = cblk->height - 1; y >= 0; y--)
//		{
//			for(x = cblk->width - 1; x >= 0; x--)
//			{
//				printf("%d, ", buff[x + y * tile_comp->cblk_w]);
//			}
//		}
//		printf("\n");
//	}
//
////	println_end(INFO);
//
//	return sb;
//}

/*void quantization_dbg(type_data *idata, int size, int w, int h)
{
	type_data *buff = (type_data*)malloc(size);
	int x = 0, y = 0;

	cuda_memcpy_dth(idata, buff, size);

	for(y = h - 1; y >= 0; y--)
	{
		for(x = w - 1; x >= 0; x--)
		{
			printf("%f, ", buff[x + y * w]);
		}
	}
	printf("\n");
	free(buff);
}*/
